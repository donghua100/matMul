#include <getopt.h>
#include "utils.cuh"

static struct option long_options[] = {
    {"help", no_argument, 0, 'h'},
    {"kernel", required_argument, 0, 'k'},
    {"size", required_argument, 0, 's'},
    {0, 0, 0, 0}
};

void usage(char *s) {
    printf("Uasge %s [OPTIONS]\n", s);
    printf("Options:\n");
    printf("  -h, --help        Display this help message\n");
    printf("  -k, --kernel      Specify a kernel\n");
    printf("  -s, --size        Specify size\n");

}

static char kernelName[][20] = {
    "cublas",
    "Naive",
    "Sm",
    "SmReg"
};

int main(int argc, char *argv[]) {
    if (argc == 1) {
        usage(argv[0]);
        exit(EXIT_FAILURE);
    }
    int opt;
    int n = 0;
    int M, N, K;
    while ((opt = getopt_long(argc, argv, "hk:s:", long_options, NULL)) != -1) {
        switch(opt) {
            case 'h':
                usage(argv[0]);
                exit(EXIT_SUCCESS);
            case 'k':
                n = atoi(optarg);
                break;
            case 's':
                int x;
                if ((x = sscanf(optarg, "%d %d %d", &M, &N, &K)) != 3) {
                    printf("optarg = %s \n", optarg);
                    printf("x = %d\n", x);
                    fprintf(stderr, "Invalid format for -s option.\n");
                    exit(EXIT_FAILURE);
                }
                break;
            case '?':
                exit(EXIT_FAILURE);
            default:
                break;
                
        }
    }

    srand(time(NULL));
    float *Ah, *Bh, *Ch, *Dh, *Eh, *Ad, *Bd, *Cd;
    Ah = (float *)malloc(sizeof(float)*M*K);
    Bh = (float *)malloc(sizeof(float)*K*N);
    Ch = (float *)malloc(sizeof(float)*M*N);
    Dh = (float *)malloc(sizeof(float)*M*N);
    Eh = (float *)malloc(sizeof(float)*M*N);

    randomMat(Ah, M*K);
    randomMat(Bh, K*N);

    hipMalloc((void **)&Ad, sizeof(float)*M*K);
    hipMalloc((void **)&Bd, sizeof(float)*K*N);
    hipMalloc((void **)&Cd, sizeof(float)*M*N);
    hipMemcpy(Ad, Ah, sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(Bd, Bh, sizeof(float)*K*N, hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    if (n != 0) {
        runKernel(0, Ad, Bd, Cd, M, N, K, handle);
        hipMemcpy(Ch, Cd, sizeof(float)*M*N, hipMemcpyDeviceToHost); 
        runKernel(n, Ad, Bd, Cd, M, N, K, handle);
        hipMemcpy(Dh, Cd, sizeof(float)*M*N, hipMemcpyDeviceToHost);
        cmpMat(Ch, Dh, M*N);
    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    int repat = 10;
    for (int i = 0; i < repat; i++) {
        runKernel(0, Ad, Bd, Cd, M, N, K, handle);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float millisecond;
    hipEventElapsedTime(&millisecond, start, end);
    printf("[%-10s]     Time    %.2f ms, Throughput     %.2f GFLOPs,    [100.00%%] cublas\n",
           kernelName[0], millisecond/repat, 2.0*M*N*K*repat/(millisecond*1e6));

    hipEventRecord(start);
    for (int i = 0; i < repat; i++) {
        runKernel(n, Ad, Bd, Cd, M, N, K, handle);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float cublas_millsecond = millisecond;
    hipEventElapsedTime(&millisecond, start, end);
    printf("[%-10s]     Time    %.2f ms, Throughput     %.2f GFLOPs,    [%.2f%%] cublas\n",
           kernelName[n], millisecond/repat, 2.0*M*N*K*repat/(millisecond*1e6), 100*cublas_millsecond/millisecond);

    free(Ah);
    free(Bh);
    free(Ch);
    free(Dh);
    free(Eh);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    hipblasDestroy(handle);
    return 0;
}

